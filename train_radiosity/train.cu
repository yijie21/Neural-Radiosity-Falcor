#include "hip/hip_runtime.h"
#include <tiny-cuda-nn/common_device.h>
#include <tiny-cuda-nn/config.h>
#define TINYEXR_IMPLEMENTATION
#include <tinyexr/tinyexr.h>
#include <filesystem/path.h>
#include <filesystem/directory.h>
#include <fstream>
#include <iostream>
#include <string>
#include <vector>

using namespace tcnn;
using precision_t = network_precision_t;

struct TrainingMetadata {
	GPUMemory<float> posW;
	GPUMemory<float> normalW;
	GPUMemory<float> wiW;
	GPUMemory<float> diff;
	GPUMemory<float> color;

	ivec2 resolution = ivec2(0);

	TrainingMetadata(const uint32_t width, const uint32_t height) {
		resolution.x = width;
		resolution.y = height;
		posW.resize(width * height * 4 * 4);	// 4 channels * 4 bytes
		normalW.resize(width * height * 4 * 4);
		wiW.resize(width * height * 4 * 4);
		diff.resize(width * height * 4 * 4);
		color.resize(width * height * 4 * 4);
	}
};

struct TrainingTexture {
	hipTextureObject_t posW;
	hipTextureObject_t normalW;
	hipTextureObject_t wiW;
	hipTextureObject_t diff;
	hipTextureObject_t color;
};

GPUMemory<float> load_image(const filesystem::path& path, int& width, int& height) {
	// width * height * RGBA
	float* out;
	const char* err = nullptr;
	int ret = LoadEXR(&out, &width, &height, path.str().c_str(), &err);
	FreeEXRErrorMessage(err);
	GPUMemory<float> result(width * height * 4);
	result.copy_from_host(out);
	free(out);
	return result;
}

TrainingMetadata load_metadata(filesystem::path& folder_path) {
	std::vector<filesystem::path> paths;
	for (auto& path: filesystem::directory(folder_path)) {
		if (path.is_file() && path.extension() == "exr") {
			paths.push_back(path);
		}
	}
	std::sort(paths.begin(), paths.end(), [](const filesystem::path& a, const filesystem::path& b) {
		return a.str() < b.str();
	});

	int width, height;

	load_image(paths[0], width, height);

	TrainingMetadata result(width, height);

	uint32_t img_size = width * height * 4 * 4;

	for (auto& path: paths) {
		size_t lastDot = path.str().rfind('.');
		size_t secondLastDot = path.str().rfind('.', lastDot - 1);
		size_t thirdLastDot = path.str().rfind('.', secondLastDot - 1);
		std::string buffer_name = path.str().substr(thirdLastDot + 1, secondLastDot - thirdLastDot - 1);

		if (buffer_name == "posW") {
			hipMemcpy(result.posW.data(), load_image(path, width, height).data(), img_size, hipMemcpyDeviceToDevice);
		}
		if (buffer_name == "normalW") {
			hipMemcpy(result.normalW.data(), load_image(path, width, height).data(), img_size, hipMemcpyDeviceToDevice);
		}
		if (buffer_name == "wiW") {
			hipMemcpy(result.wiW.data(), load_image(path, width, height).data(), img_size, hipMemcpyDeviceToDevice);
		}
		if (buffer_name == "diff") {
			hipMemcpy(result.diff.data(), load_image(path, width, height).data(), img_size, hipMemcpyDeviceToDevice);
		}
		if (buffer_name == "color") {
			hipMemcpy(result.color.data(), load_image(path, width, height).data(), img_size, hipMemcpyDeviceToDevice);
		}
	}
	return result;
}

void create_cuda_texture(GPUMemory<float>& image, uint32_t width, uint32_t height, hipTextureObject_t& texture) {
	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypePitch2D;
	resDesc.res.pitch2D.devPtr = image.data();
	resDesc.res.pitch2D.desc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
	resDesc.res.pitch2D.width = width;
	resDesc.res.pitch2D.height = height;
	resDesc.res.pitch2D.pitchInBytes = width * 4 * sizeof(float);

	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.filterMode = hipFilterModeLinear;
	texDesc.normalizedCoords = true;
	texDesc.addressMode[0] = hipAddressModeClamp;
	texDesc.addressMode[1] = hipAddressModeClamp;
	texDesc.addressMode[2] = hipAddressModeClamp;

	CUDA_CHECK_THROW(hipCreateTextureObject(&texture, &resDesc, &texDesc, nullptr));
}

TrainingTexture create_training_texture(TrainingMetadata& metadata) {
	TrainingTexture result;
	create_cuda_texture(metadata.posW, metadata.resolution.x, metadata.resolution.y, result.posW);
	create_cuda_texture(metadata.normalW, metadata.resolution.x, metadata.resolution.y, result.normalW);
	create_cuda_texture(metadata.wiW, metadata.resolution.x, metadata.resolution.y, result.wiW);
	create_cuda_texture(metadata.diff, metadata.resolution.x, metadata.resolution.y, result.diff);
	create_cuda_texture(metadata.color, metadata.resolution.x, metadata.resolution.y, result.color);

	return result;
}

void destroyTexture(TrainingTexture texture) {
	hipDestroyTextureObject(texture.posW);
	hipDestroyTextureObject(texture.normalW);
	hipDestroyTextureObject(texture.wiW);
	hipDestroyTextureObject(texture.diff);
	hipDestroyTextureObject(texture.color);
}

template <typename T, uint32_t input_stride, uint32_t output_stride>
__global__ void sample_input_output(uint32_t n_elements, TrainingTexture texture,
									T* xs_and_ys, T* input, T* output) {
	uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= n_elements) return;

	uint32_t texture_offset = i * 2;
	uint32_t input_offset = i * input_stride;
	uint32_t output_offset = i * output_stride;

	float4 posW_val 	= tex2D<float4>(texture.posW, xs_and_ys[texture_offset + 0], xs_and_ys[texture_offset + 1]);
	float4 normalW_val 	= tex2D<float4>(texture.normalW, xs_and_ys[texture_offset + 0], xs_and_ys[texture_offset + 1]);
	float4 wiW_val 		= tex2D<float4>(texture.wiW, xs_and_ys[texture_offset + 0], xs_and_ys[texture_offset + 1]);
	float4 diff_val 	= tex2D<float4>(texture.diff, xs_and_ys[texture_offset + 0], xs_and_ys[texture_offset + 1]);
	float4 color_val 	= tex2D<float4>(texture.diff, xs_and_ys[texture_offset + 0], xs_and_ys[texture_offset + 1]);

	input[input_offset + 0] = posW_val.x;		input[input_offset + 1] = posW_val.y;		input[input_offset + 2] = posW_val.z;
	// input[input_offset + 3] = normalW_val.x;	input[input_offset + 4] = normalW_val.y;	input[input_offset + 5] = normalW_val.z;
	// input[input_offset + 6] = wiW_val.x;		input[input_offset + 7] = wiW_val.y;		input[input_offset + 8] = wiW_val.z;
	// input[input_offset + 9] = diff_val.x;		input[input_offset + 10] = diff_val.y;		input[input_offset + 11] = diff_val.z;

	output[output_offset + 0] = color_val.x;	output[output_offset + 1] = color_val.y;	output[output_offset + 2] = color_val.z;
}

int main(int argc, char* argv[]) {
	try {
		uint32_t compute_capability = cuda_compute_capability();
		if (compute_capability < MIN_GPU_ARCH) {
			std::cerr
				<< "Warning: Insufficient compute capability " << compute_capability << " detected. "
				<< "This program was compiled for >=" << MIN_GPU_ARCH << " and may thus behave unexpectedly." << std::endl;
		}

		if (argc != 3) {
			std::cerr << "Usage: " << argv[0] << " <dataset folder> <json path>" << std::endl;
			return EXIT_FAILURE;
		}

		filesystem::path dataset_folder(argv[1]);
		std::vector<filesystem::path> frame_folders;
		if (!dataset_folder.empty()) {
			if (dataset_folder.is_directory()) {
				for (const auto& path: filesystem::directory(dataset_folder)) {
					if (path.is_directory() && path.str().find("frame") != std::string::npos) {
						frame_folders.push_back(path);
					}
				}
			}
		}

		if (frame_folders.empty()) {
			std::cerr << "Error: No frame folders found in dataset folder." << std::endl;
			return EXIT_FAILURE;
		}

		std::ifstream f{argv[2]};
		json config = json::parse(f, nullptr, true, true);

		const uint32_t n_training_steps = 5000;
		const uint32_t n_input_dims = 12;
		const uint32_t n_texture_dims = 2;
		const uint32_t n_output_dims = 3;
		const uint32_t batch_size = 1 << 12;
		const uint32_t n_frames = frame_folders.size();

		const uint32_t log_interval = 100;
		float tmp_loss = 0;
		uint32_t tmp_loss_counter = 0;

		hipStream_t inference_stream;
		CUDA_CHECK_THROW(hipStreamCreate(&inference_stream));
		hipStream_t training_stream = inference_stream;

		default_rng_t rng{1337};

		json encoding_opts = config.value("encoding", json::object());
		json loss_opts = config.value("loss", json::object());
		json optimizer_opts = config.value("optimizer", json::object());
		json network_opts = config.value("network", json::object());

		std::shared_ptr<Loss<precision_t>> loss{create_loss<precision_t>(loss_opts)};
		std::shared_ptr<Optimizer<precision_t>> optimizer{create_optimizer<precision_t>(optimizer_opts)};
		std::shared_ptr<NetworkWithInputEncoding<precision_t>> network = std::make_shared<NetworkWithInputEncoding<precision_t>>(n_input_dims, n_output_dims, encoding_opts, network_opts);

		auto trainer = std::make_shared<Trainer<float, precision_t, precision_t>>(network, optimizer, loss);

		GPUMatrix<float> training_input(n_input_dims, batch_size);
		GPUMatrix<float> training_output(n_output_dims, batch_size);
		GPUMatrix<float> sample_coords(n_texture_dims, batch_size);

		for (uint32_t i = 0; i < n_training_steps; i++) {
			uint32_t training_set_idx = (uint32_t)(rng.next_float() * n_frames);
			TrainingMetadata metadata = load_metadata(frame_folders[training_set_idx]);
			TrainingTexture training_texture = create_training_texture(metadata);

			generate_random_uniform<float>(training_stream, rng, batch_size * n_texture_dims, sample_coords.data());
			linear_kernel(sample_input_output<float, n_input_dims, n_output_dims>, 0, training_stream,
				batch_size, training_texture, sample_coords.data(), training_input.data(), training_output.data());

			auto ctx = trainer->training_step(training_stream, training_input, training_output);

			tmp_loss += trainer->loss(training_stream, *ctx);
			tmp_loss_counter++;

			if (i % log_interval == 0) {
				std::cout << "Step#" << i << ": " << "loss=" << tmp_loss/(float)tmp_loss_counter << std::endl;

				tmp_loss = 0;
				tmp_loss_counter = 0;
			}

			destroyTexture(training_texture);
		}

		json network_config;
		std::string network_config_save_path = "network_weights.json";
		network_config = trainer->serialize(false);
		std::ofstream of(network_config_save_path);
		of << network_config.dump(4);
		of.close();

		free_all_gpu_memory_arenas();

	} catch (const std::exception& e) {
		std::cout << "Uncaught exception: " << e.what() << std::endl;
	}

	return EXIT_SUCCESS;
}
