#include "hip/hip_runtime.h"
#include "Network.h"

#include <fstream>
#include <iostream>
#include <filesystem/path.h>

#include <tiny-cuda-nn/common_device.h>
#include <tiny-cuda-nn/config.h>
#include <json/json.hpp>

using namespace tcnn;
using precision_t = network_precision_t;

namespace
{

struct NetworkComponents {
    std::shared_ptr<Loss<precision_t>> loss = nullptr;
    std::shared_ptr<Optimizer<precision_t>> optimizer = nullptr;
    std::shared_ptr<NetworkWithInputEncoding<precision_t>> network = nullptr;
    std::shared_ptr<Trainer<float, precision_t, precision_t>> trainer = nullptr;
};

struct IOData {
    GPUMatrix<float>* input_mat = nullptr;
    GPUMatrix<float>* output_mat = nullptr;
};

hipStream_t inference_stream = nullptr;

NetworkComponents* mNetworkComponents = nullptr;

IOData* mIOData = nullptr;

}

template <typename T, uint32_t stride>
__global__ void formatInput(uint32_t n_elements, Falcor::RadiosityQuery* queries, T* input)
{
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n_elements) return;

    Falcor::RadiosityQuery query = queries[i];

    input[i * stride + 0] = query.posW.x;           input[i * stride + 1] = query.posW.y;           input[i * stride + 2] = query.posW.z;
    input[i * stride + 3] = query.normalW.x;        input[i * stride + 4] = query.normalW.y;        input[i * stride + 5] = query.normalW.z;
    input[i * stride + 6] = query.wiW.x;            input[i * stride + 7] = query.wiW.y;            input[i * stride + 8] = query.wiW.z;
    input[i * stride + 9] = query.diff.x;           input[i * stride + 10] = query.diff.y;          input[i * stride + 11] = query.diff.z;
}


template <typename T, uint32_t stride>
__global__ void mapToOutSurf(uint32_t n_elements, uint32_t width, T* output, hipSurfaceObject_t outSurf)
{
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n_elements) return;

    uint32_t x = i % width;
    uint32_t y = i / width;

    float4 color = { 0, 0, 0, 1 };

    color.x = output[i * stride + 0];
    color.y = output[i * stride + 1];
    color.z = output[i * stride + 2];

    surf2Dwrite(color, outSurf, x * sizeof(float4), y);
}


RadiosityNetwork::RadiosityNetwork(const uint32_t width, const uint32_t height)
{
    CUDA_CHECK_THROW(hipStreamCreate(&inference_stream));

    mNetworkComponents = new NetworkComponents();
    mIOData = new IOData();

    filesystem::path c_path(NetConfig::netConfigPath);
    if (!c_path.exists()) {
        std::cout << "Cannot find the network config!" << std::endl;
        return;
    } else {
        std::cout << "Successfully find the network config!" << std::endl;
    }

    std::ifstream f(c_path.str());
    json config = json::parse(f, nullptr, true, true);

    json encoding_opts = config.value("encoding", json::object());
	json loss_opts = config.value("loss", json::object());
	json optimizer_opts = config.value("optimizer", json::object());
	json network_opts = config.value("network", json::object());

    mNetworkComponents->loss = std::shared_ptr<Loss<precision_t>>(create_loss<precision_t>(loss_opts));
    mNetworkComponents->optimizer = std::shared_ptr<Optimizer<precision_t>>(create_optimizer<precision_t>(optimizer_opts));
    mNetworkComponents->network = std::make_shared<NetworkWithInputEncoding<precision_t>>(NetConfig::n_input_dims, NetConfig::n_output_dims, encoding_opts, network_opts);
    mNetworkComponents->trainer = std::make_shared<Trainer<float, precision_t, precision_t>>(mNetworkComponents->network, mNetworkComponents->optimizer, mNetworkComponents->loss);

    filesystem::path w_path(NetConfig::weightsPath);
    if (!w_path.exists()) {
        std::cout << "Cannot find the weights!" << std::endl;
        return;
    } else {
        std::cout << "Successfully find the weights!" << std::endl;
    }
    std::ifstream wf(w_path.str());
    json loaded_weights = json::parse(wf, nullptr, true, true);

    mNetworkComponents->trainer->deserialize(loaded_weights);

    mIOData->input_mat = new GPUMatrix<float>(NetConfig::n_input_dims, width * height);
    mIOData->output_mat = new GPUMatrix<float>(NetConfig::n_output_dims, width * height);

    frame_width = width;
    frame_height = height;
}


RadiosityNetwork::~RadiosityNetwork()
{
    delete mNetworkComponents;
    delete mIOData;
}


void RadiosityNetwork::forward(Falcor::RadiosityQuery* queries, hipSurfaceObject_t output)
{
    uint32_t n_elements = frame_width * frame_height;

    linear_kernel(formatInput<float, NetConfig::n_input_dims>, 0, inference_stream, n_elements, queries, mIOData->input_mat->data());

    mNetworkComponents->network->inference(inference_stream, *mIOData->input_mat, *mIOData->output_mat);

    linear_kernel(mapToOutSurf<float, NetConfig::n_output_dims>, 0, inference_stream, n_elements, frame_width, mIOData->output_mat->data(), output);
}
